extern "C" {


#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void vecAdd(int n, double* x, double* y)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   for (int idx = i; idx < n; idx += blockDim.x * gridDim.x) {
      y[idx] += x[idx];
      if (idx==0) printf("after modif, y[%d] = %f\n", idx, y[idx]);
   }
}

void vecAdd_wrapper(int n, double *x, double *y) {
  vecAdd<<<256,256>>>(n,x,y);
  hipDeviceSynchronize();
}

} //extern "C"
